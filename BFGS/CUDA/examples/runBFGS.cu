// Author: Dai-Ni Hsieh (dnhsieh@jhu.edu)
// Date  : 05/20/2020

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include "struct.h"

long long assignStructMemory(optdata &, fcndata &, double *, double *);
void      BFGS(double *, double *, double *, optdata &, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	mxGPUArray const *posIni;
	mxGPUArray       *posNow, *grdNow;

	optdata optObj;
	fcndata fcnObj;

	posIni        = mxGPUCreateFromMxArray(prhs[0]);
	optObj.itrMax =            mxGetScalar(prhs[1]);
	optObj.tolVal =            mxGetScalar(prhs[2]);
	optObj.wolfe1 =            mxGetScalar(prhs[3]);
	optObj.wolfe2 =            mxGetScalar(prhs[4]);
	optObj.vbsFlg =            mxGetScalar(prhs[5]);

	fcnObj.varNum = mxGPUGetNumberOfElements(posIni);

	if ( fcnObj.varNum > 1000000 )
	{
		mexErrMsgIdAndTxt("BFGS:numberOfVariables", 
		                  "The number of variables is over one million. Use LBFGS instead.");
	}

	mwSize const ndim = 1;
	mwSize const dims[1] = {(mwSize) fcnObj.varNum};
	posNow = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	grdNow = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

	// ---

	// inputs
	double *d_posIni = (double *) mxGPUGetDataReadOnly(posIni);

	// outputs
	double *d_posNow = (double *) mxGPUGetData(posNow);
	double  h_fcnNow;
	double *d_grdNow = (double *) mxGPUGetData(grdNow);

	// ---

	long long optMemCnt = (long long) fcnObj.varNum * (fcnObj.varNum + 6);
	long long fcnMemCnt = 0;

	double *d_optWorkspace = NULL, *d_fcnWorkspace = NULL;
	hipError_t optMemErr = hipMalloc((void **) &d_optWorkspace, sizeof(double) * optMemCnt);
	hipError_t fcnMemErr = hipMalloc((void **) &d_fcnWorkspace, sizeof(double) * fcnMemCnt);

	if ( optMemErr != hipSuccess || fcnMemErr != hipSuccess )
	{
		mexErrMsgIdAndTxt("BFGS:memAllocate", 
		                  "Failed to allocate memory for BFGS.");
	}

	long long asgMemCnt = assignStructMemory(optObj, fcnObj, d_optWorkspace, d_fcnWorkspace);
	if ( asgMemCnt != optMemCnt + fcnMemCnt )
	{
		mexErrMsgIdAndTxt("BFGS:memAssign", 
		                  "Assigned memory (%d) mismatched the allocated memory (%d).", 
		                  asgMemCnt, optMemCnt + fcnMemCnt);
	}

	hipblasCreate(&(fcnObj.blasHdl));

	// ---

	hipMemcpy(d_posNow, d_posIni, sizeof(double) * fcnObj.varNum, hipMemcpyDeviceToDevice);
	BFGS(&h_fcnNow, d_grdNow, d_posNow, optObj, fcnObj);

	plhs[0] = mxGPUCreateMxArrayOnGPU(  posNow);
	plhs[1] =    mxCreateDoubleScalar(h_fcnNow);
	plhs[2] = mxGPUCreateMxArrayOnGPU(  grdNow);

	// ---

	mxGPUDestroyGPUArray(posIni);
	mxGPUDestroyGPUArray(posNow);
	mxGPUDestroyGPUArray(grdNow);

	hipFree(d_optWorkspace);
	hipFree(d_fcnWorkspace);

	hipblasDestroy(fcnObj.blasHdl);

	return;
}

