#include "hip/hip_runtime.h"
#include "struct.h"

// global minimum  : 0
// global minimizer: (1, 1, ..., 1)

__global__ void objgrdKernel(double *d_fcnVal, double *d_grdVec, double *d_posVec, int varNum)
{
	*d_fcnVal = 0.0;
	for ( int varIdx = 0; varIdx < varNum - 1; ++varIdx )
	{
		double x0 = d_posVec[varIdx    ];
		double x1 = d_posVec[varIdx + 1];
		double f0 = x1 - x0 * x0;
		double f1 = 1.0 - x0;
		
		*d_fcnVal += 100.0 * f0 * f0 + f1 * f1;

		d_grdVec[varIdx    ] += -400.0 * f0 * x0 - 2.0 * f1;
		d_grdVec[varIdx + 1] +=  200.0 * f0;
	}

	return;
}

void objgrd(double *h_fcnVal, double *d_grdVec, double *d_posVec, fcndata &fcnObj)
{
	int varNum = fcnObj.varNum;

	hipMemset(d_grdVec, 0, sizeof(double) * varNum);

	// bad implementation, but enough for the purpose of illustration
	double *d_fcnVal;
	hipMalloc((void **) &d_fcnVal, sizeof(double));
	objgrdKernel <<<1, 1>>> (d_fcnVal, d_grdVec, d_posVec, varNum);

	hipMemcpy(h_fcnVal, d_fcnVal, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_fcnVal);

	return;
}
