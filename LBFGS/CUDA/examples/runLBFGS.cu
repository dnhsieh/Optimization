// Author: Dai-Ni Hsieh (dnhsieh@jhu.edu)
// Date  : 05/20/2020

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include "struct.h"

long long assignStructMemory(optdata &, fcndata &, double *, double *);
void      LBFGS(double *, double *, double *, optdata &, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	mxGPUArray const *posIni;
	mxGPUArray       *posNow, *grdNow;

	optdata optObj;
	fcndata fcnObj;

	posIni        = mxGPUCreateFromMxArray(prhs[0]);
	optObj.memNum =            mxGetScalar(prhs[1]);
	optObj.itrMax =            mxGetScalar(prhs[2]);
	optObj.tolVal =            mxGetScalar(prhs[3]);
	optObj.wolfe1 =            mxGetScalar(prhs[4]);
	optObj.wolfe2 =            mxGetScalar(prhs[5]);
	optObj.vbsFlg =            mxGetScalar(prhs[6]);

	fcnObj.varNum = mxGPUGetNumberOfElements(posIni);

	mwSize const ndim = 1;
	mwSize const dims[1] = {(mwSize) fcnObj.varNum};
	posNow = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	grdNow = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

	// ---

	// inputs
	double *d_posIni = (double *) mxGPUGetDataReadOnly(posIni);

	// outputs
	double *d_posNow = (double *) mxGPUGetData(posNow);
	double  h_fcnNow;
	double *d_grdNow = (double *) mxGPUGetData(grdNow);

	// ---

	long long optMemCnt = (long long) fcnObj.varNum * (5 + 2 * optObj.memNum);
	long long fcnMemCnt = 0;

	double *d_optWorkspace = NULL, *d_fcnWorkspace = NULL;
	hipError_t optMemErr = hipMalloc((void **) &d_optWorkspace, sizeof(double) * optMemCnt);
	hipError_t fcnMemErr = hipMalloc((void **) &d_fcnWorkspace, sizeof(double) * fcnMemCnt);

	if ( optMemErr != hipSuccess || fcnMemErr != hipSuccess )
	{
		mexErrMsgIdAndTxt("LBFGS:memAllocate", 
		                  "Failed to allocate memory for LBFGS.");
	}

	long long asgMemCnt = assignStructMemory(optObj, fcnObj, d_optWorkspace, d_fcnWorkspace);
	if ( asgMemCnt != optMemCnt + fcnMemCnt )
	{
		mexErrMsgIdAndTxt("LBFGS:memAssign", 
		                  "Assigned memory (%d) mismatched the allocated memory (%d).", 
		                  asgMemCnt, optMemCnt + fcnMemCnt);
	}

	optObj.h_recVec = (double *) calloc(optObj.memNum, sizeof(double));

	hipblasCreate(&(fcnObj.blasHdl));

	// ---

	hipMemcpy(d_posNow, d_posIni, sizeof(double) * fcnObj.varNum, hipMemcpyDeviceToDevice);
	LBFGS(&h_fcnNow, d_grdNow, d_posNow, optObj, fcnObj);

	plhs[0] = mxGPUCreateMxArrayOnGPU(  posNow);
	plhs[1] =    mxCreateDoubleScalar(h_fcnNow);
	plhs[2] = mxGPUCreateMxArrayOnGPU(  grdNow);

	// ---

	mxGPUDestroyGPUArray(posIni);
	mxGPUDestroyGPUArray(posNow);
	mxGPUDestroyGPUArray(grdNow);

	free(optObj.h_recVec);
	hipFree(d_optWorkspace);
	hipFree(d_fcnWorkspace);

	hipblasDestroy(fcnObj.blasHdl);

	return;
}

