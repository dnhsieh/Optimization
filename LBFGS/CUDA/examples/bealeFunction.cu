#include "hip/hip_runtime.h"
#include "struct.h"

// global minimum  : 0
// global minimizer: (3, 0.5)

__global__ void objgrdKernel(double *d_fcnVal, double *d_grdVec, double *d_posVec, int varNum)
{
	double xVal = d_posVec[0];
	double yVal = d_posVec[1];

	double f1Val = 1.5   + xVal * (-1.0 + yVal              );
	double f2Val = 2.25  + xVal * (-1.0 + yVal * yVal       );
	double f3Val = 2.625 + xVal * (-1.0 + yVal * yVal * yVal);

	*d_fcnVal = f1Val * f1Val + f2Val * f2Val + f3Val * f3Val;

	d_grdVec[0] =  2 * f1Val * (-1.0 + yVal              )
	             + 2 * f2Val * (-1.0 + yVal * yVal       )
	             + 2 * f3Val * (-1.0 + yVal * yVal * yVal);

	d_grdVec[1] =  2 * f1Val *       xVal
	             + 2 * f2Val * 2.0 * xVal * yVal
	             + 2 * f3Val * 3.0 * xVal * yVal * yVal;

	return;
}

void objgrd(double *h_fcnVal, double *d_grdVec, double *d_posVec, fcndata &fcnObj)
{
	int varNum = fcnObj.varNum;

	hipMemset(d_grdVec, 0, sizeof(double) * varNum);

	// bad implementation, but enough for the purpose of illustration
	double *d_fcnVal;
	hipMalloc((void **) &d_fcnVal, sizeof(double));
	objgrdKernel <<<1, 1>>> (d_fcnVal, d_grdVec, d_posVec, varNum);

	hipMemcpy(h_fcnVal, d_fcnVal, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_fcnVal);


	return;
}
